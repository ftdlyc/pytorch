#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <cmath>

/* Fake quantize a tensor, common block for per-channel & per-tensor fake quant
Args:
  output: output tensor.
  input : input tensor.
  sc:  scale to quantize the input tensor to
  zero_point: zero_point
  quant_min: minimum quantized value
  quant_max: maximum quantized value
Returns:
  Fake quantized tensor (double dtype).
*/
namespace at {
namespace native {
void fake_quantize_slice_cuda(
    Tensor& output,
    const Tensor& input,
    float scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max,
    int64_t rounding_method) {
  float inv_scale = 1.0f / scale;
  switch(rounding_method) {
  case 0:
    at::cuda::CUDA_tensor_apply2<float, float>(
      input, output, [=] __device__(const float& input_val, float& result_val) {
        result_val = (fminf(
                          quant_max,
                          fmaxf(
                              quant_min,
                              static_cast<int64_t>(std::nearbyint(
                                  input_val * inv_scale + zero_point)))) -
                      zero_point) *
            scale;
      });
    break;
  case 1:
    at::cuda::CUDA_tensor_apply2<float, float>(
      input, output, [=] __device__(const float& input_val, float& result_val) {
        result_val = (fminf(
                          quant_max,
                          fmaxf(
                              quant_min,
                              static_cast<int64_t>(std::round(
                                  input_val * inv_scale + zero_point)))) -
                      zero_point) *
            scale;
      });
    break;
  case 2:
    at::cuda::CUDA_tensor_apply2<float, float>(
      input, output, [=] __device__(const float& input_val, float& result_val) {
        result_val = (fminf(
                          quant_max,
                          fmaxf(
                              quant_min,
                              static_cast<int64_t>(std::floor(
                                  input_val * inv_scale + zero_point)))) -
                      zero_point) *
            scale;
      });
    break;
  case 3:
    at::cuda::CUDA_tensor_apply2<float, float>(
      input, output, [=] __device__(const float& input_val, float& result_val) {
        result_val = (fminf(
                          quant_max,
                          fmaxf(
                              quant_min,
                              static_cast<int64_t>(std::ceil(
                                  input_val * inv_scale + zero_point)))) -
                      zero_point) *
            scale;
      });
    break;
  case 4:
    at::cuda::CUDA_tensor_apply2<float, float>(
      input, output, [=] __device__(const float& input_val, float& result_val) {
        result_val = (fminf(
                          quant_max,
                          fmaxf(
                              quant_min,
                              static_cast<int64_t>(std::trunc(
                                  input_val * inv_scale + zero_point)))) -
                      zero_point) *
            scale;
      });
    break;
  default:
    at::cuda::CUDA_tensor_apply2<float, float>(
      input, output, [=] __device__(const float& input_val, float& result_val) {
        result_val = (fminf(
                          quant_max,
                          fmaxf(
                              quant_min,
                              static_cast<int64_t>(std::nearbyint(
                                  input_val * inv_scale + zero_point)))) -
                      zero_point) *
            scale;
      });
  }
}

void fake_quantize_grad_slice_cuda(
    Tensor& input_grad,
    const Tensor& input,
    const Tensor& output_grad,
    float scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max,
    int64_t rounding_method) {
  float inv_scale = 1.0f / scale;
  switch(rounding_method) {
  case 0:
    at::cuda::CUDA_tensor_apply3<float, float, float>(
      output_grad,
      input,
      input_grad,
      [=] __device__(const float& dy, const float& x, float& dx) {
        int64_t Xq = std::nearbyint(x * inv_scale + zero_point);
        dx = (Xq >= quant_min && Xq <= quant_max) * dy;
      });
    break;
  case 1:
    at::cuda::CUDA_tensor_apply3<float, float, float>(
      output_grad,
      input,
      input_grad,
      [=] __device__(const float& dy, const float& x, float& dx) {
        int64_t Xq = std::round(x * inv_scale + zero_point);
        dx = (Xq >= quant_min && Xq <= quant_max) * dy;
      });
    break;
  case 2:
    at::cuda::CUDA_tensor_apply3<float, float, float>(
      output_grad,
      input,
      input_grad,
      [=] __device__(const float& dy, const float& x, float& dx) {
        int64_t Xq = std::floor(x * inv_scale + zero_point);
        dx = (Xq >= quant_min && Xq <= quant_max) * dy;
      });
    break;
  case 3:
    at::cuda::CUDA_tensor_apply3<float, float, float>(
      output_grad,
      input,
      input_grad,
      [=] __device__(const float& dy, const float& x, float& dx) {
        int64_t Xq = std::ceil(x * inv_scale + zero_point);
        dx = (Xq >= quant_min && Xq <= quant_max) * dy;
      });
    break;
  case 4:
    at::cuda::CUDA_tensor_apply3<float, float, float>(
      output_grad,
      input,
      input_grad,
      [=] __device__(const float& dy, const float& x, float& dx) {
        int64_t Xq = std::trunc(x * inv_scale + zero_point);
        dx = (Xq >= quant_min && Xq <= quant_max) * dy;
      });
    break;
  default:
    at::cuda::CUDA_tensor_apply3<float, float, float>(
      output_grad,
      input,
      input_grad,
      [=] __device__(const float& dy, const float& x, float& dx) {
        int64_t Xq = std::nearbyint(x * inv_scale + zero_point);
        dx = (Xq >= quant_min && Xq <= quant_max) * dy;
      });
  }
}

} // namespace native
} // namespace at
